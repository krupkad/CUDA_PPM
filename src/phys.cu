#include "hip/hip_runtime.h"
#include "ppm.hpp"
#include "bezier.hpp"
#include "util/error.hpp"

#include <glm/gtc/type_ptr.hpp>

#include <iterator>
#include <algorithm>

__global__ void kCalcInertia(int nFace, const int4 *heFaces, const float *vtxData, glm::mat3 *moiOut, float *massOut, glm::vec3 *cmOut) {
  int fIdx = threadIdx.x + blockIdx.x * blockDim.x;
  if (fIdx >= nFace)
    return;
    
  extern __shared__ glm::mat3 matSM[];
  glm::mat3 &C = matSM[0];
  glm::mat3 &A = matSM[1+threadIdx.x];
  float *pA = glm::value_ptr(A);
  float *pC = glm::value_ptr(C);
  
  for (int i = 0; i < 3; i++) {
    for (int j = 0; j < 3; j++) {
      pA[3*i + j] = vtxData[PPM_NVARS*heFaces[3*fIdx + i].x + j];
      if (threadIdx.x == 0) pC[3*i + j] = (i == j) ? (1.0f/60) : (1.0f/120);
    }
  }
  
  float detA = glm::determinant(A);
  float *cmPtr = glm::value_ptr(*cmOut);
  atomicAdd(&cmPtr[fIdx+0], detA*(pA[0]+pA[3]+pA[6])/18.0f);
  atomicAdd(&cmPtr[fIdx+1], detA*(pA[1]+pA[4]+pA[7])/18.0f);
  atomicAdd(&cmPtr[fIdx+2], detA*(pA[2]+pA[5]+pA[8])/18.0f);
  atomicAdd(massOut, detA/6);
  
  __syncthreads();
  float *moiPtr = glm::value_ptr(*moiOut);
  A = detA * A * C * glm::transpose(A);
  for (int i = 0; i < 9; i++)
	  atomicAdd(&moiPtr[i], pA[i]);
}

__global__ void kCalcTessInertia(int nFace, const int *tessIdx, const float *tessVtx, glm::mat3 *moiOut, float *massOut, glm::vec3 *cmOut) {
  int fIdx = threadIdx.x + blockIdx.x * blockDim.x;
  if (fIdx >= nFace)
    return;
    
  extern __shared__ glm::mat3 matSM[];
  glm::mat3 &C = matSM[0];
  glm::mat3 &A = matSM[1+threadIdx.x];
  float *pA = glm::value_ptr(A);
  float *pC = glm::value_ptr(C);
  
  for (int i = 0; i < 3; i++) {
    for (int j = 0; j < 3; j++) {
      pA[3*i + j] = tessVtx[PPM_NVARS*tessIdx[3*fIdx + i] + j];
      if (threadIdx.x == 0) pC[3*i + j] = (i == j) ? (1.0f/60) : (1.0f/120);
    }
  }
  
  float detA = glm::determinant(A);
  float *cmPtr = glm::value_ptr(*cmOut);
  atomicAdd(&cmPtr[fIdx+0], detA*(pA[0]+pA[3]+pA[6])/18.0f);
  atomicAdd(&cmPtr[fIdx+1], detA*(pA[1]+pA[4]+pA[7])/18.0f);
  atomicAdd(&cmPtr[fIdx+2], detA*(pA[2]+pA[5]+pA[8])/18.0f);
  atomicAdd(massOut, detA/6);
  
  __syncthreads();
  float *moiPtr = glm::value_ptr(*moiOut);
  A = detA * A * C * glm::transpose(A);
  for (int i = 0; i < 9; i++)
	atomicAdd(&moiPtr[i], pA[i]);
}

__global__ void kCenter(int nVtx, glm::vec3 cm, float *vData) {
  int vIdx = threadIdx.x + blockDim.x * blockIdx.x;
  if (vIdx >= nVtx)
    return;
  
  vData[PPM_NVARS*vIdx + 0] -= cm[0];
  vData[PPM_NVARS*vIdx + 1] -= cm[1];
  vData[PPM_NVARS*vIdx + 2] -= cm[2];
}

void PPM::physInit() {
  fprintf(stderr, "phys alloc\n");
  glm::mat3 *dev_moi;
  hipMalloc(&dev_moi, sizeof(glm::mat3));
  hipMemset(dev_moi, 0, sizeof(glm::mat3));
  float *dev_mass;
  hipMalloc(&dev_mass, sizeof(float));
  hipMemset(dev_mass, 0, sizeof(float));
  glm::vec3 *dev_cm;
  hipMalloc(&dev_cm, sizeof(glm::vec3));
  hipMemset(dev_cm, 0, sizeof(glm::vec3));
  
  fprintf(stderr, "phys compute\n");
  dim3 blkDim(256), blkCnt((nFace + 255)/256);
  int nSM = (1+blkDim.x) * sizeof(glm::mat3);
  kCalcInertia<<<blkCnt,blkDim,nSM>>>(nFace, dev_heFaces, dev_vList, dev_moi, dev_mass, dev_cm);
  
  hipMemcpy(&moi, dev_moi, sizeof(glm::mat3), hipMemcpyDeviceToHost);
  hipMemcpy(&cm, dev_cm, sizeof(glm::vec3), hipMemcpyDeviceToHost);
  hipMemcpy(&mass, dev_mass, sizeof(float), hipMemcpyDeviceToHost);

  kCenter<<<blkCnt,blkDim>>>(nVtx, cm, dev_vList);
  hipMemcpy(&vList[0], dev_vList, PPM_NVARS*nVtx*sizeof(float), hipMemcpyDeviceToHost);
  
  fprintf(stderr, "phys reduce\n");
  float tr = moi[0][0] + moi[1][1] + moi[2][2];
  for (int i = 0; i < 3; i++) {
  for (int j = 0; j < 3; j++) {
    moi[i][j] = ((i == j) ? tr : 0.0f) - moi[i][j];
  }}

  fprintf(stderr, "phys result: %f (%f %f %f)\n", mass, cm.x, cm.y, cm.z);
  fprintf(stderr, "%f %f %f\n", moi[0][0], moi[1][0], moi[2][0]);
  fprintf(stderr, "%f %f %f\n", moi[0][1], moi[1][1], moi[2][1]);
  fprintf(stderr, "%f %f %f\n\n", moi[0][2], moi[1][2], moi[2][2]);
  
  hipFree(dev_moi);
  hipFree(dev_mass);
  hipFree(dev_cm);
}

void PPM::physTess() {
  fprintf(stderr, "phys alloc\n");
  glm::mat3 *dev_moi;
  hipMalloc(&dev_moi, sizeof(glm::mat3));
  hipMemset(dev_moi, 0, sizeof(glm::mat3));
  float *dev_mass;
  hipMalloc(&dev_mass, sizeof(float));
  hipMemset(dev_mass, 0, sizeof(float));
  glm::vec3 *dev_cm;
  hipMalloc(&dev_cm, sizeof(glm::vec3));
  hipMemset(dev_cm, 0, sizeof(glm::vec3));
  
  fprintf(stderr, "phys compute\n");
  dim3 blkDim(256), blkCnt((nFace + 255)/256);
  int nSM = (1+blkDim.x) * sizeof(glm::mat3);
  kCalcTessInertia<<<blkCnt,blkDim,nSM>>>(nFace, dev_tessIdx, dev_tessVtx, dev_moi, dev_mass, dev_cm);
  
  hipMemcpy(&moi, dev_moi, sizeof(glm::mat3), hipMemcpyDeviceToHost);
  hipMemcpy(&cm, dev_cm, sizeof(glm::vec3), hipMemcpyDeviceToHost);
  hipMemcpy(&mass, dev_mass, sizeof(float), hipMemcpyDeviceToHost);

  fprintf(stderr, "phys reduce\n");
  moi -= mass*glm::outerProduct(cm,cm);
  moi = glm::mat3(moi[0][0] + moi[1][1] + moi[2][2]) - moi;
  fprintf(stderr, "phys result: %f (%f %f %f)\n", mass, cm.x, cm.y, cm.z);
  fprintf(stderr, "%f %f %f\n", moi[0][0], moi[1][0], moi[2][0]);
  fprintf(stderr, "%f %f %f\n", moi[0][1], moi[1][1], moi[2][1]);
  fprintf(stderr, "%f %f %f\n\n", moi[0][2], moi[1][2], moi[2][2]);
  
  hipFree(dev_moi);
  hipFree(dev_mass);
  hipFree(dev_cm);
}

__global__ void kMeshIntersect(bool exec, bool biDir,
                                int nSubFace, const int *vTessIdx, const float *vTessData,
                                const glm::vec3 p0, const glm::vec3 dir,
                                unsigned int *count, int *idxOut, float *tOut) {
  int fSubIdx = threadIdx.x + blockIdx.x * blockDim.x;
  if (fSubIdx >= nSubFace)
    return;
  
  vTessIdx = &vTessIdx[3*fSubIdx];
  const float *v0 = &vTessData[PPM_NVARS*vTessIdx[0]];
  const float *v1 = &vTessData[PPM_NVARS*vTessIdx[1]];
  const float *v2 = &vTessData[PPM_NVARS*vTessIdx[2]];
  
  float e1[3], e2[3];
  for (int i = 0; i < 3; i++) {
    e1[i] = v1[i] - v0[i];
    e2[i] = v2[i] - v0[i];
  }

  float p[3];
  p[0] = dir[1]*e2[2] - dir[2]*e2[1];
  p[1] = dir[2]*e2[0] - dir[0]*e2[2];
  p[2] = dir[0]*e2[1] - dir[1]*e2[0];
  
  float idet = e1[0]*p[0] + e1[1]*p[1] + e1[2]*p[2];
  if (idet > -1e-5 && idet < 1e-5)
    return;
  idet = 1.0f/idet;
  
  float T[3];
  for (int i = 0; i < 3; i++)
    T[i] = p0[i] - v0[i];
  float u = idet*(p[0]*T[0] + p[1]*T[1] + p[2]*T[2]);
  if (u < 0 || u > 1)
    return;
  
  
  p[0] = T[1]*e1[2] - T[2]*e1[1];
  p[1] = T[2]*e1[0] - T[0]*e1[2];
  p[2] = T[0]*e1[1] - T[1]*e1[0];
  float v = idet*(dir[0]*p[0] + dir[1]*p[1] + dir[2]*p[2]);
  if (v < 0 || u+v > 1)
    return;
  
  float t = idet*(e2[0]*p[0] + e2[1]*p[1] + e2[2]*p[2]);
  if (biDir || (t > 1e-5)) {
    if (exec) {
      int oIdx = atomicSub(count, 1) - 1;
      tOut[oIdx] = t;
      idxOut[oIdx] = fSubIdx;
    } else {
      atomicAdd(count, 1);
    }
  }
}

__global__ void kUpdateCoeff(int nBasis2, int nVtx, const float *V, float sigma, const float *dv, float *coeff, float dt) {
  int bIdx = threadIdx.x + blockIdx.x * blockDim.x;
  int vIdx = threadIdx.y + blockIdx.y * blockDim.y;
  if (vIdx >= nVtx || bIdx >= nBasis2)
    return;

  dv = &dv[9*vIdx];

  float v = sigma * V[0*nBasis2 + bIdx];
  int tIdx = bIdx + vIdx*nBasis2;
  coeff[tIdx + 0 * nVtx*nBasis2] += dv[3] * v * dt;
  coeff[tIdx + 1 * nVtx*nBasis2] += dv[4] * v * dt;
  coeff[tIdx + 2 * nVtx*nBasis2] += dv[5] * v * dt;
}

__global__ void kPhysVerlet1(int nVtx, float *dv, float rMass, const int2 *vBndList, float dt) {
  int vIdx = threadIdx.x + blockIdx.x * blockDim.x;
  if (vIdx >= nVtx)
    return;

  const int2 &bnd = vBndList[vIdx];
  //rMass *= bnd.y - bnd.x;

  dv = &dv[9*vIdx];
  dv[3] += 0.5f*dv[6]*dt / rMass;
  dv[4] += 0.5f*dv[7]*dt / rMass;
  dv[5] += 0.5f*dv[8]*dt / rMass;
  dv[0] += dv[3]*dt;
  dv[1] += dv[4]*dt;
  dv[2] += dv[5]*dt;

  dv[6] = 0.0f;
  dv[7] = 0.0f;
  dv[8] = 0.0f;
}

__global__ void kPhysNeighbor(int nHe, const int4 *heLoops, float kSelf, float kDamp, float kNbr, float *dv) {
  int heIdx = threadIdx.x + blockIdx.x * blockDim.x;
  if (heIdx >= nHe)
    return;

  const int4 &he = heLoops[heIdx];
  kSelf = kSelf + kNbr;
  atomicAdd(&dv[9*he.x + 6], kNbr * dv[9*he.y + 0] - kSelf * dv[9*he.x + 0] - kDamp*dv[9*he.x + 3]); 
  atomicAdd(&dv[9*he.x + 7], kNbr * dv[9*he.y + 1] - kSelf * dv[9*he.x + 1] - kDamp*dv[9*he.x + 4]);
  atomicAdd(&dv[9*he.x + 8], kNbr * dv[9*he.y + 2] - kSelf * dv[9*he.x + 2] - kDamp*dv[9*he.x + 5]);
}

__global__ void kPhysNeighborAlt(int nVtx, const int4 *heLoops, const int2 *vBndList,
                                 float kSelf, float kDamp, float kNbr, float *dv) {
  int vIdx = threadIdx.x + blockIdx.x * blockDim.x;
  if (vIdx >= nVtx)
    return;
  const int2 &bnd = vBndList[vIdx];

  extern __shared__ float physSM[];
  float *vSM = &physSM[3 * threadIdx.x];
  kSelf = -(kSelf + kNbr*(bnd.y - bnd.x));
  vSM[0] = kSelf * dv[9*vIdx + 0];
  vSM[1] = kSelf * dv[9*vIdx + 1];
  vSM[2] = kSelf * dv[9*vIdx + 2];

  //kNbr /= (bnd.y - bnd.x);
  for (int i = bnd.x; i < bnd.y; i++) {
    int tgt = heLoops[i].y;
    vSM[0] += kNbr * dv[9*tgt + 0];
    vSM[1] += kNbr * dv[9*tgt + 1];
    vSM[2] += kNbr * dv[9*tgt + 2];
  }

  dv[9*vIdx + 6] = vSM[0] - kDamp*dv[9*vIdx + 3];
  dv[9*vIdx + 7] = vSM[1] - kDamp*dv[9*vIdx + 4];
  dv[9*vIdx + 8] = vSM[2] - kDamp*dv[9*vIdx + 5];
}

__global__ void kPhysVerlet2(int nVtx, float *dv, float dt) {
  int vIdx = threadIdx.x + blockIdx.x * blockDim.x;
  if (vIdx >= nVtx)
    return;

  dv = &dv[9*vIdx];
  dv[3] += 0.5f*dv[6]*dt;
  dv[4] += 0.5f*dv[7]*dt;
  dv[5] += 0.5f*dv[8]*dt;
}


/*
  
  if (uv.x > 0 && uv.y > 0) {
    fSubIdx = fIdx*nSubFace + UV_IDX(uv.x - 1, uv.y - 1) + nSubVtx - nSub - 1;
    idxOut[3*fSubIdx + 0] = tessGetIdx(uv.x, uv.y, heLoops, heFaces,  heTessOrder, fIdx, nVtx, nHe, nFace, nSub);
    idxOut[3*fSubIdx + 1] = tessGetIdx(uv.x-1, uv.y, heLoops, heFaces,  heTessOrder, fIdx, nVtx, nHe, nFace, nSub);
    idxOut[3*fSubIdx + 2] = tessGetIdx(uv.x, uv.y-1, heLoops, heFaces,  heTessOrder, fIdx, nVtx, nHe, nFace, nSub);
  }

  if (uv.x+uv.y < nSub) {
    fSubIdx = fIdx*nSubFace + UV_IDX(uv.x, uv.y);
    idxOut[3*fSubIdx + 0] = tessGetIdx(uv.x, uv.y, heLoops, heFaces,  heTessOrder, fIdx, nVtx, nHe, nFace, nSub);
    idxOut[3*fSubIdx + 1] = tessGetIdx(uv.x+1, uv.y, heLoops, heFaces,  heTessOrder, fIdx, nVtx, nHe, nFace, nSub);
    idxOut[3*fSubIdx + 2] = tessGetIdx(uv.x, uv.y + 1, heLoops, heFaces, heTessOrder, fIdx, nVtx, nHe, nFace, nSub);
  }
*/
__global__ void kPhysClick(int nSub, int nSubVtx, int fSubIdx, const float2 *uvIdx, 
                            const int4 *heFaces, const float *vData,
                            float force, float *dv) {
  int fIdx = fSubIdx / (nSub*nSub);
  int uvOff = fSubIdx - fIdx*nSub*nSub;

  float2 uv;
  if (uvOff >= nSubVtx - nSub - 1) {
    uv = uvIdx[uvOff - (nSubVtx - nSub - 1)];
    uv.x += 1.0f/nSub;
    uv.y += 1.0f/nSub;
  } else {
    uv = uvIdx[uvOff];
  }
  float w = 1.0f - uv.x - uv.y;

  const int4 &he0 = heFaces[3*fIdx], &he1 = heFaces[3*fIdx+1], &he2 = heFaces[3*fIdx+2];
  const float *v0 = &vData[PPM_NVARS*he0.x], *v1 = &vData[PPM_NVARS*he1.x], *v2 = &vData[PPM_NVARS*he2.x];
  float *dv0 = &dv[9*he0.x], *dv1 = &dv[9*he1.x], *dv2 = &dv[9*he2.x];

  float dx = force;
  dv0[6] += w * dx * v0[3];
  dv0[7] += w * dx * v0[4];
  dv0[8] += w * dx * v0[5];
  dv1[6] += uv.x * dx * v1[3];
  dv1[7] += uv.x * dx * v1[4];
  dv1[8] += uv.x * dx * v1[5];
  dv2[6] += uv.y * dx * v2[3];
  dv2[7] += uv.y * dx * v2[4];
  dv2[8] += uv.y * dx * v2[5];
}

int PPM::intersect(const glm::vec3 &p0, const glm::vec3 &dir) {
  if (!isBuilt)
    return false;

  unsigned int *dev_count;
  hipMalloc(&dev_count, sizeof(unsigned int));
  hipMemset(dev_count, 0, sizeof(unsigned int));
  dim3 blkCnt((nFace*nSubFace + 255) / 256), blkDim(256);
  kMeshIntersect<<<blkCnt,blkDim>>>(false, false, nFace*nSubFace, dev_tessIdx, dev_tessVtx, p0, dir, dev_count, nullptr, nullptr);
  checkCUDAError("kMeshIntersect", __LINE__);
  
  unsigned int count;
  hipMemcpy(&count, dev_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
  if (!count) {
    //printf("no ix\n");
    hipFree(dev_count);
    return -1;
  }
  
  float2 *dev_uvOut;
  float *dev_tOut;
  int *dev_idxOut;
  hipMalloc(&dev_tOut, count*sizeof(float));
  hipMalloc(&dev_idxOut, count*sizeof(int));
  kMeshIntersect<<<blkCnt,blkDim>>>(true, false, nFace*nSubFace, dev_tessIdx, dev_tessVtx, p0, dir, dev_count, dev_idxOut, dev_tOut);
  checkCUDAError("kMeshIntersect", __LINE__);
  
  std::vector<float> tOut(count);
  std::vector<int> idxOut(count);
  hipMemcpy(&tOut[0], dev_tOut, count*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&idxOut[0], dev_idxOut, count*sizeof(int), hipMemcpyDeviceToHost);
  
  int minPos = std::min_element(tOut.begin(), tOut.end()) - tOut.begin();
  int idx = idxOut[minPos];

  //printf("ix %f %f\n", uv.x, uv.y);
  hipFree(dev_tOut);
  hipFree(dev_idxOut);
  return idx;
}

void PPM::updateCoeff(int clickIdx,float clickForce,  float dt) {
  dim3 blkDim(128), blkCnt;
  blkCnt.x = (nVtx + blkDim.x - 1) / blkDim.x;
  kPhysVerlet1<<<blkCnt,blkDim>>>(nVtx, dev_dv, 1000.0f*mass/nVtx, dev_vBndList, dt);
  checkCUDAError("kPhysVerlet1", __LINE__); 
  
  blkDim.x = 16;
  blkDim.y = 64;
  blkCnt.x = (nBasis2 + blkDim.x - 1) / blkDim.x;
  blkCnt.y = (nVtx + blkDim.y - 1) / blkDim.y;
  kUpdateCoeff<<<blkCnt,blkDim>>>(nBasis2, nVtx, bezier->dev_V, 1.0, dev_dv, dev_coeff, dt);
  checkCUDAError("kUpdateCoeff", __LINE__);

  blkDim.x = 256;
  blkDim.y = 1;
  blkCnt.x = (nVtx + blkDim.x - 1) / blkDim.x;
  blkCnt.y = 1;
  int nSM = 3*blkDim.x*sizeof(float);
  kPhysNeighborAlt<<<blkCnt,blkDim,nSM>>>(nVtx, dev_heLoops, dev_vBndList, kSelf, kDamp, kNbr, dev_dv);
  if (clickIdx > -1)
    kPhysClick<<<1,1>>>(nSub, nSubVtx, clickIdx, dev_uvIdxMap, dev_heFaces, dev_vList, clickForce, dev_dv);
  //kPhysNeighbor<<<blkCnt,blkDim>>>(nHe, dev_heLoops, kSelf, kDamp, kNbr, dev_dv);
  checkCUDAError("kPhysNeighbor", __LINE__);
  
  blkCnt.x = (nVtx + blkDim.x - 1) / blkDim.x;
  kPhysVerlet2<<<blkCnt,blkDim>>>(nVtx, dev_dv, dt);
  checkCUDAError("kPhysVerlet2", __LINE__); 
}

