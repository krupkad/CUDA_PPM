#include "hip/hip_runtime.h"
#include "ppm.hpp"

#include <glm/gtc/type_ptr.hpp>

//#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>

__global__ void kCalcInertia(int nFace, const int4 *heFaces, const float *vtxData, glm::mat3 *moiOut, float *massOut, glm::vec3 *cmOut) {
  int fIdx = threadIdx.x + blockIdx.x * blockDim.x;
  if (fIdx >= nFace)
    return;
    
  extern __shared__ glm::mat3 matSM[];
  glm::mat3 &C = matSM[0];
  glm::mat3 &A = matSM[1+threadIdx.x];
  float *pA = glm::value_ptr(A);
  float *pC = glm::value_ptr(C);
  
  for (int i = 0; i < 3; i++) {
    for (int j = 0; j < 3; j++) {
      pA[3*i + j] = vtxData[PPM_NVARS*heFaces[3*fIdx + i].x + j];
      if (fIdx == 0) pC[3*i + j] = (i == j) ? (1.0f/60) : (1.0f/120);
    }
  }
  float detA = glm::determinant(A);
  
  cmOut[fIdx] = detA*(A[0] + A[1] + A[2])/18.0f;
  massOut[fIdx] = detA/6;
  __syncthreads();
  moiOut[fIdx] = detA * A * C * glm::transpose(A);
}

void PPM::physInit() {
  fprintf(stderr, "phys alloc\n");
  glm::mat3 *dev_moi;
  hipMalloc(&dev_moi, nFace*sizeof(glm::mat3));
  float *dev_mass;
  hipMalloc(&dev_mass, nFace*sizeof(float));
  glm::vec3 *dev_cm;
  hipMalloc(&dev_cm, nFace*sizeof(glm::vec3));
  
  fprintf(stderr, "phys compute\n");
  dim3 blkDim(256), blkCnt((nFace + 255)/256);
  int nSM = (1+blkDim.x) * sizeof(glm::mat3);
  kCalcInertia<<<blkCnt,blkDim,nSM>>>(nFace, dev_heFaces, dev_vList, dev_moi, dev_mass, dev_cm);
  
  thrust::device_ptr<float> ptr_mass(dev_mass);
  thrust::device_ptr<glm::mat3> ptr_moi(dev_moi);
  thrust::device_ptr<glm::vec3> ptr_cm(dev_cm);

  fprintf(stderr, "phys reduce\n");
  moi = thrust::reduce(ptr_moi, ptr_moi+nFace);
  mass = thrust::reduce(ptr_mass, ptr_mass+nFace);
  cm = thrust::reduce(ptr_cm, ptr_cm+nFace) / mass;
  moi -= mass*glm::outerProduct(cm,cm);
  moi = glm::mat3(moi[0][0] + moi[1][1] + moi[2][2]) - moi;
  
  hipFree(dev_moi);
  hipFree(dev_mass);
  hipFree(dev_cm);
  
  fprintf(stderr, "phys result: %f (%f %f %f)\n", mass, cm.x, cm.y, cm.z);
  fprintf(stderr, "%f %f %f\n", moi[0][0], moi[1][0], moi[2][0]);
  fprintf(stderr, "%f %f %f\n", moi[0][1], moi[1][1], moi[2][1]);
  fprintf(stderr, "%f %f %f\n\n", moi[0][2], moi[1][2], moi[2][2]);
}

