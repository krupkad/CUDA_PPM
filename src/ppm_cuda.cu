#include "hip/hip_runtime.h"
#include "ppm.hpp"
#include "bezier.hpp"

#include <thrust/iterator/counting_iterator.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

// output weights and bezier basis coefficients for tessellated subvertices of a patch
__global__ void kBezEval(int deg, int nBasis, int nSubVtx, const float2 *uvIdxMap, float *bzOut, float *wgtOut) {
  int dIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int uvIdx = blockIdx.y * blockDim.y + threadIdx.y;
  if (dIdx >= deg || uvIdx >= nSubVtx)
    return;

  // each thread needs nBasis*sizeof(float2) to compute all basis functions
  // plus 2*sizeof(float2) to store p and (1-p)
  extern __shared__ float2 sAll[];
  int nSM = (nBasis + 2);
  int tIdx = threadIdx.x + blockDim.x * threadIdx.y;
  float2 &p = sAll[tIdx*nSM + 0];
  float2 &np = sAll[tIdx*nSM + 1];
  float2 *sWork = &sAll[tIdx*nSM + 2];

  // get sector-local xy
  const float2 &uv = uvIdxMap[uvIdx];
  float a = 2.0 / deg;
  float ca, sa;
  sincospif(a, &sa, &ca);
  float x = uv.x + uv.y*ca, y = uv.y*sa;
  a *= dIdx;
  sincospif(a, &sa, &ca);

  // calculate bernstein polynomials
  p = 0.5f*make_float2(x*ca - y*sa, x*sa + y*ca) + 0.5f;
  np = 1.0f - p;
  kBnBasis<float>(nBasis, p, np, sWork);

  // compute weight
  float h2 = cospif(1.0f / (deg > 4 ? deg : 4)), h1 = 0.25f*h2;
  float r = hypotf(x, y);
  float h = (r - h1) / (h2 - h1);
  float s = rsqrtf(1.0f - h) - rsqrtf(h);
  float w = (r < h1) ? 1.0f : ((r > h2) ? 0.0f :( 1.0f / (1.0f + expf(2.0f*s))));

  // tensor product and output
  int oIdx = dIdx*nSubVtx + uvIdx;
  for (int k = 0; k < nBasis; k++) {
  for (int j = 0; j < nBasis; j++) {
    bzOut[j + nBasis*k + oIdx*nBasis*nBasis] = sWork[j].x * sWork[k].y;
  }}
  wgtOut[oIdx] = w;
}

// given a sorted list, find indices of where each block starts/ends
__global__ void kGetLoopBoundaries(int nHe, const int4 *heList, int2 *vBndList) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= nHe)
    return;

  if (i == 0 || heList[i-1].x != heList[i].x)
    vBndList[heList[i].x].x = i;
  if (i == nHe - 1 || heList[i + 1].x != heList[i].x)
    vBndList[heList[i].x].y = i+1;
}

// given loop boundaries, fill in he.z (loop order) and he.w (loop degree)
__global__ void kGetHeRootInfo(int nHe, const int2 *vBndList, int4 *heList, int4 *heLoops) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= nHe)
    return;

  int4 &he = heList[i];
  const int2 &bnd = vBndList[he.x];
  he.w = bnd.y - bnd.x;
  he.z = -1;
  for (int j = bnd.x; j < bnd.y; j++) {
    if (heLoops[j].y == he.y) {
      he.z = j - bnd.x;
      return;
    }
  }
}

__global__ void kMeshSample(int nVert, int nGrid, int degMin,
                            hipTextureObject_t sampTexObj,
                            const int4 *heLoops, const int2 *vHeLoopBnd,
                            const float *vData, float *samp) {
  int x = blockIdx.x * blockDim.x + threadIdx.x; // grid idx
  int y = blockIdx.y * blockDim.y + threadIdx.y; // grid idx
  int vtxIdx = blockIdx.z * blockDim.z + threadIdx.z; // vert idx
  if (x >= nGrid || y >= nGrid || vtxIdx >= nVert)
    return;

  const int2 &heBnd = vHeLoopBnd[vtxIdx];
  float4 uvi = tex2DLayered<float4>(sampTexObj, x, y, heBnd.y - heBnd.x - degMin);
  float u = uvi.x, v = uvi.y, w = 1.0f - u - v;
  int heOff = uvi.z;

  const int4 &he0 = heLoops[heBnd.x + heOff];
  const int4 &he1 = heLoops[heBnd.x + (heOff + 1)%he0.w];
  int sIdx = x + nGrid*y + vtxIdx*nGrid*nGrid, sDim = nGrid*nGrid*nVert;
  const float *p0, *p1, *p2;

  p0 = &vData[PPM_NVARS * he0.x];
  p1 = &vData[PPM_NVARS * he0.y];
  p2 = &vData[PPM_NVARS * he1.y];
  for (int i = 0; i < PPM_NVARS; i++)
    samp[sIdx + i * sDim] = p0[i] * w + p1[i] * u + p2[i] * v;
}

__global__ void kMeshSampleOrig(int nVert, int nGrid, int degMin,
                                hipTextureObject_t sampTexObj,
                                const int4 *heLoops, const int2 *vHeLoopBnd,
                                const float *vtx, float *samp) {
  int ix = blockIdx.x * blockDim.x + threadIdx.x; // grid idx
  int iy = blockIdx.y * blockDim.y + threadIdx.y; // grid idx
  int vtxIdx = blockIdx.z * blockDim.z + threadIdx.z; // vert idx
  if (ix >= nGrid || iy >= nGrid || vtxIdx >= nVert)
    return;
  float x = 2.0f * float(ix) / (nGrid-1) - 1.0f;
  float y = 2.0f * float(iy) / (nGrid-1) - 1.0f;

  const int2 &heBnd = vHeLoopBnd[vtxIdx];
  int deg = heBnd.y - heBnd.x;
  float alpha = 2.0f*M_PI / deg;
  float th = ((y < 0) ? 2.0f*M_PI : 0.0f) + atan2f(y, x);
  float r = hypotf(x, y);
  float dTh = fmodf(th, alpha);
  int ord = floorf(th / alpha);
  float v = r*sinf(dTh) / sinf(alpha);
  float u = r*cosf(dTh) - v*cosf(alpha);
  float k = u + v;
  if (k > 1.0) {
	  u /= k;
	  v /= k;
  }
  float w = 1.0 - u - v;

  const int4 &he0 = heLoops[heBnd.x + ord];
  const int4 &he1 = heLoops[heBnd.x + (ord + 1) % he0.w];
  int sIdx = ix + nGrid*iy + vtxIdx*nGrid*nGrid, sDim = nGrid*nGrid*nVert;
  const float *p0, *p1, *p2;

  p0 = &vtx[PPM_NVARS * he0.x];
  p1 = &vtx[PPM_NVARS * he0.y];
  p2 = &vtx[PPM_NVARS * he1.y];
  for (int i = 0; i < PPM_NVARS; i++)
    samp[sIdx + i * sDim] = p0[i] * w + p1[i] * u + p2[i] * v;
}

__global__ void kGetHeTessInfo(int nHe, int degMin, const int4 *heLoops, int4 *heTarg, const int2 *vBndList) {
  int heIdx = threadIdx.x + blockIdx.x * blockDim.x;
  if (heIdx >= nHe)
    return;
  
  int4 &he = heTarg[heIdx];
  he.z = he.z + (he.w*(he.w - 1) - degMin*(degMin - 1)) / 2;
  
  he.w = -1;
  const int2 &bnd = vBndList[he.y];
  for (int i = bnd.x; i < bnd.y; i++) {
    if (heLoops[i].y == he.x) {
      he.w = i;
      break;
    }
  }
}

__global__ void kGetHeTessIdx(int nHe, const int4 *heLoops, int *heTessIdx) {
  int heIdx = threadIdx.x + blockIdx.x * blockDim.x;
  if (heIdx >= nHe)
    return;

  if (heIdx < heLoops[heIdx].w)
    heTessIdx[heIdx] = -heIdx-1;
  else
    heTessIdx[heIdx] = heLoops[heIdx].w+1;
}

__device__ inline float patchContrib(int dIdx, int vIdx, int nBasis2, const float *bez, const float *wgt, const float *coeff, float &res) {
  bez = &bez[dIdx*nBasis2];
  float w = wgt[dIdx];
  for (int i = 0; i < nBasis2; i++)
    res += w * bez[i] * coeff[i + vIdx * nBasis2];
  return w;
}

// generate the per-face template for tessellation
#define UV_IDX(u,v) (((u)+(v)+1)*((u)+(v))/2 + (v))
__global__ void kTessVtx_Face(int nVtx, int nFace, int nSub, int nBasis2,
                        const int4 *heFaces, const float *bezData, const float *wgtData, const float *coeff,
                        const int2 *uvIdxMap, float *vDataOut) {
  int fIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int uvIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int dataIdx = blockIdx.z * blockDim.z + threadIdx.z;
  int nSubVtx = (nSub+1)*(nSub+2)/2;
  if (fIdx >= nFace || uvIdx >= (nSub-1)*(nSub-2)/2 || dataIdx >= PPM_NVARS)
    return;
    
  const int2 &uv = uvIdxMap[uvIdx];
  const int4 &he0 = heFaces[3 * fIdx + 0];
  const int4 &he1 = heFaces[3 * fIdx + 1];
  const int4 &he2 = heFaces[3 * fIdx + 2];

  float res = 0.0, wgt = 0.0;
  wgt += patchContrib(he0.z*nSubVtx + UV_IDX(uv.x, uv.y), he0.x + dataIdx*nVtx, nBasis2, bezData, wgtData, coeff, res);
  wgt += patchContrib(he1.z*nSubVtx + UV_IDX(uv.y, nSub-uv.x-uv.y), he1.x + dataIdx*nVtx, nBasis2, bezData, wgtData, coeff, res);
  wgt += patchContrib(he2.z*nSubVtx + UV_IDX(nSub-uv.x-uv.y, uv.x), he2.x + dataIdx*nVtx, nBasis2, bezData, wgtData, coeff, res);

  vDataOut[PPM_NVARS*(fIdx*(nSub-2)*(nSub-1)/2 + UV_IDX(uv.x-1,uv.y-1)) + dataIdx] = res / wgt;
}

__global__ void kTessVtx_Edge(int nVtx, int nHe, int nSub, int nBasis2,
                        const int4 *heLoops, const float *bezData, const float *wgtData, const float *coeff,
                        float *vDataOut) {
  int heIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int uIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int dataIdx = blockIdx.z * blockDim.z + threadIdx.z;
  if (heIdx >= nHe || uIdx >= nSub-1 || dataIdx >= PPM_NVARS)
    return;

  const int4 &he0 = heLoops[heIdx];
  const int4 &he1 = heLoops[he0.w];
  if (heIdx >= he0.w)
    return;

  float res = 0.0, wgt = 0.0;
  int nSubVtx = (nSub+1)*(nSub+2)/2;
  wgt += patchContrib(he0.z*nSubVtx + UV_IDX(uIdx+1,0), he0.x + dataIdx*nVtx, nBasis2, bezData, wgtData, coeff, res);
  wgt += patchContrib(he1.z*nSubVtx + UV_IDX(nSub-uIdx-1,0), he1.x + dataIdx*nVtx, nBasis2, bezData, wgtData, coeff, res);

  vDataOut[PPM_NVARS*(heIdx*(nSub-1) + uIdx) + dataIdx] = res / wgt;
}

__global__ void kTessVtx_Edge_Alt(int nVtx, int nHe, int nSub, int nBasis2,
                        const int4 *heLoops, const int *heTessOrder, 
                        const float *bezData, const float *wgtData, const float *coeff,
                        float *vDataOut) {
  int heIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int uIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int dataIdx = blockIdx.z * blockDim.z + threadIdx.z;
  if (heIdx >= nHe || uIdx >= nSub-1 || dataIdx >= PPM_NVARS)
    return;

  const int4 &he0 = heLoops[heIdx];
  const int4 &he1 = heLoops[he0.w];
  if (heIdx >= he0.w)
    return;

  float res = 0.0, wgt = 0.0;
  int nSubVtx = (nSub+1)*(nSub+2)/2;
  wgt += patchContrib(he0.z*nSubVtx + UV_IDX(uIdx+1,0), he0.x + dataIdx*nVtx, nBasis2, bezData, wgtData, coeff, res);
  wgt += patchContrib(he1.z*nSubVtx + UV_IDX(nSub-uIdx-1,0), he1.x + dataIdx*nVtx, nBasis2, bezData, wgtData, coeff, res);

  vDataOut[PPM_NVARS*(heTessOrder[heIdx]*(nSub-1) + uIdx) + dataIdx] = res / wgt;
}


__global__ void kTessVtx_Vtx(int nVtx, int nSub,  int nBasis2,
                        const int4 *heLoops, const int2 *vBndList, 
                        const float *bezData, const float *wgtData, const float *coeff,
                        float *vDataOut) {
  int vIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int dataIdx = blockIdx.y * blockDim.y + threadIdx.y;
  if (vIdx >= nVtx || dataIdx >= PPM_NVARS)
    return;

  const int4 &he = heLoops[vBndList[vIdx].x];
  int dIdx = he.z*(nSub+1)*(nSub+2)/2;
  
  const float *bez = &bezData[dIdx*nBasis2];
  float wgt = wgtData[dIdx], res;
  for (int i = 0; i < nBasis2; i++)
    res += wgt * bez[i] * coeff[i + (he.x + dataIdx * nVtx) * nBasis2];

  vDataOut[PPM_NVARS*vIdx + dataIdx] = res / wgt;
}

__global__ void kWeightScale(int nFace, int nSubVtx, float *vData, float *wgt) {
  int fIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int uvIdx = blockIdx.y * blockDim.y + threadIdx.y;
  if (fIdx >= nFace || uvIdx >= nSubVtx)
    return;

  float w = wgt[fIdx*nSubVtx + uvIdx];

  for (int i = 0; i < PPM_NVARS; i++)
    vData[PPM_NVARS * (fIdx*nSubVtx + uvIdx) + i] /= w;
}

__global__ void kTessVtxSM(int nVtx, int nHe, int nSub, int nSubVtx, int nBasis2, int degMin,
  const int4 *heFaces, const float *bezData, const float *wgtData, const float *coeff,
  const int2 *uvIdxMap, float *vtxOut, float *wgtOut) {
  int heIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int uvIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int dataIdx = blockIdx.z * blockDim.z + threadIdx.z;
  if (heIdx >= nHe || dataIdx >= PPM_NVARS)
    return;

  const int4 &he = heFaces[heIdx];
  int dOff = he.z + (he.w*(he.w - 1) - degMin*(degMin - 1)) / 2;

  extern __shared__ float sTessVtxAltAll2[];
  float *sLoc = &sTessVtxAltAll2[(blockDim.x*threadIdx.z + threadIdx.x) * nBasis2];

  int fIdx = heIdx / 3;
  float *out = &vtxOut[PPM_NVARS * (fIdx*nSubVtx + uvIdx)];
  float *wgt = &wgtOut[fIdx*nSubVtx + uvIdx];

  for (int i = threadIdx.y; i < nBasis2; i += blockDim.y)
    sLoc[i] = coeff[i + he.x*nBasis2 + dataIdx*nBasis2*nVtx];
  __syncthreads();

  if (uvIdx >= nSubVtx)
    return;
  const int2 &uv = uvIdxMap[uvIdx];
  const int uvRot[4] = { uv.x, uv.y, nSub - uv.x - uv.y, uv.x };
  int uvIdxLoc = UV_IDX(uvRot[heIdx % 3], uvRot[heIdx % 3 + 1]);

  float v = 0.0, w = wgtData[dOff*nSubVtx + uvIdxLoc];
  for (int i = 0; i < nBasis2; i++)
    v += sLoc[i] * bezData[i + dOff*nSubVtx*nBasis2 + uvIdxLoc*nBasis2] * w;

  atomicAdd(&out[dataIdx], v);
  if (dataIdx == 0)
    atomicAdd(wgt, w);
}

__global__ void kGetNormals(int nHe, const int4 *heLoops, float *vData) {
  int heIdx = threadIdx.x + blockIdx.x * blockDim.x;
  if (heIdx >= nHe)
    return;

  const int4 &he0 = heLoops[heIdx];
  const int4 &he1 = heLoops[heIdx - he0.z + (he0.z + 1) % he0.w];
  float dx1[3], dx0[3];
  for (int i = 0; i < 3; i++) {
    dx1[i] = vData[PPM_NVARS * he1.y + i] - vData[PPM_NVARS * he1.x + i];
    dx0[i] = vData[PPM_NVARS * he0.y + i] - vData[PPM_NVARS * he0.x + i];
  }

  atomicAdd(&vData[PPM_NVARS * he0.x + 3], dx0[1] * dx1[2] - dx0[2] * dx1[1]);
  atomicAdd(&vData[PPM_NVARS * he0.x + 4], dx0[2] * dx1[0] - dx0[0] * dx1[2]);
  atomicAdd(&vData[PPM_NVARS * he0.x + 5], dx0[0] * dx1[1] - dx0[1] * dx1[0]);
}

__global__ void kUpdateCoeff(int nBasis2, int nSamp, float *V, float sigma, float *dv, float *coeff, float dt) {
  int bIdx = threadIdx.x + blockIdx.x * blockDim.x;
  int sIdx = threadIdx.y + blockIdx.y * blockDim.y;
  if (sIdx >= nSamp || bIdx >= nBasis2)
    return;

  float v = sigma * V[bIdx];
  int tIdx = bIdx + sIdx*nBasis2;
  coeff[tIdx + 0 * nSamp*nBasis2] += dv[6 * sIdx + 3] * v * dt;
  coeff[tIdx + 1 * nSamp*nBasis2] += dv[6 * sIdx + 4] * v * dt;
  coeff[tIdx + 2 * nSamp*nBasis2] += dv[6 * sIdx + 5] * v * dt;
  dv[6 * sIdx + 0] += dv[6 * sIdx + 3] * dt;
  dv[6 * sIdx + 1] += dv[6 * sIdx + 4] * dt;
  dv[6 * sIdx + 2] += dv[6 * sIdx + 5] * dt;
  dv[6 * sIdx + 3] -= 0.5f*dv[6 * sIdx + 0]*dt;
  dv[6 * sIdx + 4] -= 0.5f*dv[6 * sIdx + 1] * dt;
  dv[6 * sIdx + 5] -= 0.5f*dv[6 * sIdx + 2] * dt;
}

__global__ void kUpdateCoeffSM(int nBasis2, int nSamp, float *V, float sigma, float *dv, float *coeff, float dt) {
  int bIdx = threadIdx.x + blockIdx.x * blockDim.x;
  int sIdx = threadIdx.y + blockIdx.y * blockDim.y;
  if (sIdx >= nSamp || bIdx >= nBasis2)
    return;

  extern __shared__ float ucSM[];
  float *dvSM = &ucSM[6 * threadIdx.y];
  for (int i = threadIdx.x; i < 6; i += blockDim.x)
    dvSM[i] = dv[6 * sIdx + i];
  __syncthreads();


  float v = sigma * V[bIdx];
  int tIdx = bIdx + sIdx*nBasis2;
  coeff[tIdx + 0 * nSamp*nBasis2] += dvSM[0] * v * dt;
  coeff[tIdx + 1 * nSamp*nBasis2] += dvSM[1] * v * dt;
  coeff[tIdx + 2 * nSamp*nBasis2] += dvSM[2] * v * dt;
  dvSM[0] += dvSM[3] * dt;
  dvSM[1] += dvSM[4] * dt;
  dvSM[2] += dvSM[5] * dt;
  dvSM[3] -= 0.5f*dvSM[0] * dt;
  dvSM[4] -= 0.5f*dvSM[1] * dt;
  dvSM[5] -= 0.5f*dvSM[2] * dt;

  for (int i = threadIdx.x; i < 6; i += blockDim.x)
    dv[6 * sIdx + i] = dvSM[i];
  __syncthreads();
}

__device__ inline int tessGetIdx_Alt(int u, int v, const int4 *heLoops, const int4 *heFaces,
                                const int *heTessIdx, const int *heTessOrder,
                                int fIdx, int nVtx, int nHe, int nFace, int nSub) {
  const int4 &he0 = heFaces[3*fIdx+0];
  const int4 &he1 = heFaces[3*fIdx+1];
  const int4 &he2 = heFaces[3*fIdx+2];
  int heIdx1 = heTessOrder[he0.w];
  int heIdx2 = heTessOrder[he1.w];
  int heIdx3 = heTessOrder[he2.w];
  int w = nSub-u-v;
  
  if (u == 0 && v == 0)
    return nFace*(nSub-1)*(nSub-2)/2 + nHe*(nSub-1) + he0.x;
  if (w == 0 && v == 0)
    return nFace*(nSub-1)*(nSub-2)/2 + nHe*(nSub-1) + he1.x;
  if (u == 0 && w == 0)
    return nFace*(nSub-1)*(nSub-2)/2 + nHe*(nSub-1) + he2.x;
  
  if (v == 0) {
    if (heIdx1 < nHe/2)
      return nFace*(nSub-1)*(nSub-2)/2 + heIdx1*(nSub-1) + w-1;
    else
      return nFace*(nSub-1)*(nSub-2)/2 + (nHe-heIdx1-1)*(nSub-1) + u-1;
  }
  if (w == 0) {
    if (heIdx2 < nHe/2)
      return nFace*(nSub-1)*(nSub-2)/2 + heIdx2*(nSub-1) + u-1;
    else
      return nFace*(nSub-1)*(nSub-2)/2 + (nHe-heIdx2-1)*(nSub-1) + v-1;
  }
  if (u == 0) {
    if (heIdx3 < nHe/2)
      return nFace*(nSub-1)*(nSub-2)/2 + heIdx3*(nSub-1) + v-1;
    else
      return nFace*(nSub-1)*(nSub-2)/2 + (nHe-heIdx3-1)*(nSub-1) + w-1;
  }
  
  return fIdx*(nSub-1)*(nSub-2)/2 + UV_IDX(u-1,v-1);
    
}


__device__ inline int tessGetIdx(int u, int v, const int4 *heLoops, const int4 *heFaces, 
                                int fIdx, int nVtx, int nHe, int nFace, int nSub) {
  const int4 &he0 = heFaces[3*fIdx+0];
  const int4 &he1 = heFaces[3*fIdx+1];
  const int4 &he2 = heFaces[3*fIdx+2];
  int w = nSub-u-v;
  
  if (u == 0 && v == 0)
    return nFace*(nSub-1)*(nSub-2)/2 + nHe*(nSub-1) + he0.x;
  if (w == 0 && v == 0)
    return nFace*(nSub-1)*(nSub-2)/2 + nHe*(nSub-1) + he1.x;
  if (u == 0 && w == 0)
    return nFace*(nSub-1)*(nSub-2)/2 + nHe*(nSub-1) + he2.x;
  
  if (v == 0) {
    if (he0.w < heLoops[he0.w].w)
      return nFace*(nSub-1)*(nSub-2)/2 + he0.w*(nSub-1) + w-1;
    else
      return nFace*(nSub-1)*(nSub-2)/2 + heLoops[he0.w].w*(nSub-1) + u-1;
  }
  if (w == 0) {
    if (he1.w < heLoops[he1.w].w)
      return nFace*(nSub-1)*(nSub-2)/2 + he1.w*(nSub-1) + u-1;
    else
      return nFace*(nSub-1)*(nSub-2)/2 + heLoops[he1.w].w*(nSub-1) + v-1;
  }
  if (u == 0) {
    if (he2.w < heLoops[he2.w].w)
      return nFace*(nSub-1)*(nSub-2)/2 + he2.w*(nSub-1) + v-1;
    else
      return nFace*(nSub-1)*(nSub-2)/2 + heLoops[he2.w].w*(nSub-1) + w-1;
  }
  
  return fIdx*(nSub-1)*(nSub-2)/2 + UV_IDX(u-1,v-1);
    
}

__global__ void kTessEdges(int nVtx, int nHe, int nFace, int nSub, const int4 *heLoops, const int4 *heFaces, const int2 *uvIdxMap, int *idxOut) {
  int fIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int vSubIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int nSubVtx = (nSub + 1)*(nSub + 2) / 2;
  int nSubFace = nSub*nSub;
  if (fIdx >= nFace || vSubIdx >= nSubVtx)
    return;

  int fSubIdx;
  const int2 &uv = uvIdxMap[vSubIdx];
  
  if (uv.x > 0 && uv.y > 0) {
    fSubIdx = fIdx*nSubFace + UV_IDX(uv.x - 1, uv.y - 1) + nSubVtx - nSub - 1;
    idxOut[3*fSubIdx + 0] = tessGetIdx(uv.x, uv.y, heLoops, heFaces, fIdx, nVtx, nHe, nFace, nSub);
    idxOut[3*fSubIdx + 1] = tessGetIdx(uv.x-1, uv.y, heLoops, heFaces, fIdx, nVtx, nHe, nFace, nSub);
    idxOut[3*fSubIdx + 2] = tessGetIdx(uv.x, uv.y-1, heLoops, heFaces, fIdx, nVtx, nHe, nFace, nSub);
  }

  if (uv.x+uv.y < nSub) {
    fSubIdx = fIdx*nSubFace + UV_IDX(uv.x, uv.y);
    idxOut[3*fSubIdx + 0] = tessGetIdx(uv.x, uv.y, heLoops, heFaces, fIdx, nVtx, nHe, nFace, nSub);
    idxOut[3*fSubIdx + 1] = tessGetIdx(uv.x+1, uv.y, heLoops, heFaces, fIdx, nVtx, nHe, nFace, nSub);
    idxOut[3*fSubIdx + 2] = tessGetIdx(uv.x, uv.y + 1, heLoops, heFaces, fIdx, nVtx, nHe, nFace, nSub);
  }
}

__global__ void kTessEdges_Alt(int nVtx, int nHe, int nFace, int nSub,
                                const int4 *heLoops, const int4 *heFaces, const int *heTessIdx, const int *heTessOrder,
                                const int2 *uvIdxMap, int *idxOut) {
  int fIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int vSubIdx = blockIdx.y * blockDim.y + threadIdx.y;
  int nSubVtx = (nSub + 1)*(nSub + 2) / 2;
  int nSubFace = nSub*nSub;
  if (fIdx >= nFace || vSubIdx >= nSubVtx)
    return;

  int fSubIdx;
  const int2 &uv = uvIdxMap[vSubIdx];
  
  if (uv.x > 0 && uv.y > 0) {
    fSubIdx = fIdx*nSubFace + UV_IDX(uv.x - 1, uv.y - 1) + nSubVtx - nSub - 1;
    idxOut[3*fSubIdx + 0] = tessGetIdx_Alt(uv.x, uv.y, heLoops, heFaces,  heTessIdx, heTessOrder, fIdx, nVtx, nHe, nFace, nSub);
    idxOut[3*fSubIdx + 1] = tessGetIdx_Alt(uv.x-1, uv.y, heLoops, heFaces,  heTessIdx, heTessOrder, fIdx, nVtx, nHe, nFace, nSub);
    idxOut[3*fSubIdx + 2] = tessGetIdx_Alt(uv.x, uv.y-1, heLoops, heFaces,  heTessIdx, heTessOrder, fIdx, nVtx, nHe, nFace, nSub);
  }

  if (uv.x+uv.y < nSub) {
    fSubIdx = fIdx*nSubFace + UV_IDX(uv.x, uv.y);
    idxOut[3*fSubIdx + 0] = tessGetIdx_Alt(uv.x, uv.y, heLoops, heFaces,  heTessIdx, heTessOrder, fIdx, nVtx, nHe, nFace, nSub);
    idxOut[3*fSubIdx + 1] = tessGetIdx_Alt(uv.x+1, uv.y, heLoops, heFaces,  heTessIdx, heTessOrder, fIdx, nVtx, nHe, nFace, nSub);
    idxOut[3*fSubIdx + 2] = tessGetIdx_Alt(uv.x, uv.y + 1, heLoops, heFaces, heTessIdx, heTessOrder, fIdx, nVtx, nHe, nFace, nSub);
  }
}

// generate sampling pattern textures
void PPM::genSampTex() {
  fprintf(stderr, "populating patch maps (%d-%d = %d)\n", degMin, degMax, nDeg);
  float4 *sampTexData = new float4[nGrid * nGrid * nDeg];
  for (int d = 0; d < nDeg; d++) {
    for (int j = 0; j < nGrid; j++) {
      for (int i = 0; i < nGrid; i++) {
        float x(2 * i - nGrid + 1); x /= nGrid - 1;
        float y(2 * j - nGrid + 1); y /= nGrid - 1;

        float alpha = 2.0f*M_PI / (d + degMin);
        float th = ((y < 0) ? 2.0f*M_PI : 0.0f) + atan2f(y, x);
        float r = hypotf(x, y);
        float dTh = fmodf(th, alpha);
        int ord = floorf(th / alpha);
        float v = r*sinf(dTh) / sinf(alpha);
        float u = r*cosf(dTh) - v*cosf(alpha);
		    float w = 1.0f - u - v;
		    if (w < 0) {
			    float k = u + v;
			    u /= k;
			    v /= k;
			    w = 0.0f;
		    }

        sampTexData[i + j*nGrid + d*nGrid*nGrid] = make_float4(u, v, ord, 0);
      }
    }
  }

  fprintf(stderr, "allocating texture memory\n");
  dev_sampTexArray = nullptr;
  hipChannelFormatDesc channel = hipCreateChannelDesc<float4>();
  hipMalloc3DArray(&dev_sampTexArray, &channel,
    make_hipExtent(nGrid, nGrid, nDeg), hipArrayLayered);
  checkCUDAError("hipMalloc3DArray", __LINE__);

  hipMemcpy3DParms cpyParms = { 0 };
  cpyParms.srcPos = make_hipPos(0, 0, 0);
  cpyParms.dstPos = make_hipPos(0, 0, 0);
  cpyParms.srcPtr = make_hipPitchedPtr(sampTexData, nGrid*sizeof(float4), nGrid, nGrid);
  cpyParms.dstArray = dev_sampTexArray;
  cpyParms.extent = make_hipExtent(nGrid, nGrid, nDeg);
  cpyParms.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&cpyParms);
  delete sampTexData;
  checkCUDAError("hipMemcpy3D", __LINE__);

  fprintf(stderr, "creating texture\n");
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof resDesc);
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = dev_sampTexArray;

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof texDesc);
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  hipCreateTextureObject(&sampTexObj, &resDesc, &texDesc, nullptr);
  //checkCUDAError("hipCreateTextureObject", __LINE__);
}

void PPM::genCoeff() {
  dim3 blkDim;
  dim3 blkCnt;

  // generate mesh sample points
  blkDim.x = 4;
  blkDim.y = 4;
  blkDim.z = 32;
  blkCnt.x = (nGrid + blkDim.x - 1) / blkDim.x;
  blkCnt.y = (nGrid + blkDim.y - 1) / blkDim.y;
  blkCnt.z = (nVtx + blkDim.z - 1) / blkDim.z;
  if (canUseTexObjs && useSampTex) {
    kMeshSample<<<blkCnt, blkDim>>>(nVtx, nGrid, degMin,
      sampTexObj,
      dev_heLoops, dev_vBndList,
      dev_vList, dev_samp);
    checkCUDAError("kMeshSample", __LINE__);
  }
  else {
    kMeshSampleOrig<<<blkCnt, blkDim>>>(nVtx, nGrid, degMin,
      sampTexObj,
      dev_heLoops, dev_vBndList,
      dev_vList, dev_samp);
    checkCUDAError("kMeshSampleOrig", __LINE__);
  }

  bezier->getCoeff(nVtx, dev_samp, dev_coeff);
}

void PPM::devCoeffInit() {
   // allocate and generate coefficients
  devAlloc(&dev_samp, PPM_NVARS * nGrid2 * nVtx * sizeof(float));
  devAlloc(&dev_coeff, PPM_NVARS * nBasis2 * nVtx * sizeof(float));
  genCoeff();

  // initialize the deformation vector
  float *dv = new float[6 * nVtx];
  for (int i = 0; i < nVtx; i++) {
    dv[6 * i + 0] = 0.0f;
    dv[6 * i + 1] = 0.0f;
    dv[6 * i + 2] = 0.0f;
    dv[6 * i + 3] = 0.1 * (float(rand()) / RAND_MAX - 0.5f);
    dv[6 * i + 4] = 0.1 * (float(rand()) / RAND_MAX - 0.5f);
    dv[6 * i + 5] = 0.1 * (float(rand()) / RAND_MAX - 0.5f);
  }
  devAlloc(&dev_dv, 6 * nVtx*sizeof(float));
  hipMemcpy(dev_dv, dv, 6 * nVtx*sizeof(float), hipMemcpyHostToDevice);
  delete dv;
}

void PPM::devMeshInit() {
  fprintf(stderr, "uploading mesh data\n");
  devAlloc(&dev_vList, PPM_NVARS*nVtx*sizeof(float));
  hipMemcpy(dev_vList, &vList[0],  PPM_NVARS*nVtx*sizeof(float), hipMemcpyHostToDevice);
  devAlloc(&dev_heFaces, nHe*sizeof(int4));
  hipMemcpy(dev_heFaces, &heFaces[0], nHe*sizeof(int4), hipMemcpyHostToDevice);

  // populate the loops
  fprintf(stderr, "sorting loops\n");
  getHeLoops();
  devAlloc(&dev_heLoops, nHe*sizeof(int4));
  hipMemcpy(dev_heLoops, &heLoops[0], nHe*sizeof(int4), hipMemcpyHostToDevice);

  // fill in remaining halfedge data
  devAlloc(&dev_vBndList, nVtx*sizeof(int2));
  hipMemset(dev_vBndList, 0xFF, nVtx*sizeof(int2));
  dim3 blkCnt((nHe + 1024 - 1) / 1024);
  dim3 blkDim(1024);
  kGetLoopBoundaries<<<blkCnt, blkDim>>>(nHe, dev_heLoops, dev_vBndList);
  kGetHeRootInfo<<<blkCnt, blkDim>>>(nHe, dev_vBndList, dev_heLoops, dev_heLoops);
  kGetHeRootInfo<<<blkCnt, blkDim>>>(nHe, dev_vBndList, dev_heFaces, dev_heLoops);

  // recalculate normals
  kGetNormals<<<blkCnt, blkDim>>>(nHe, dev_heLoops, dev_vList);
  checkCUDAError("kGetNormals", __LINE__);
}

void PPM::devPatchInit() {
  // initialize the bezier patch calculator
  bezier = new Bezier<float>(nBasis, nGrid);

  // build the uv index map
  fprintf(stderr, "creating uv index map %d\n", nSubVtx);
  float2 *uvIdxMap = new float2[nSubVtx];
  int2 *iuvIdxMap = new int2[nSubVtx];
  for (int v = 0; v <= nSub; v++) {
  for (int u = 0; u <= nSub - v; u++) {
	  uvIdxMap[UV_IDX(u, v)] = make_float2(float(u) / nSub, float(v) / nSub);
	  iuvIdxMap[UV_IDX(u, v)] = make_int2(u,v);
  }}
  devAlloc(&dev_uvIdxMap, nSubVtx*sizeof(float2));
  hipMemcpy(dev_uvIdxMap, uvIdxMap, nSubVtx*sizeof(float2), hipMemcpyHostToDevice);
  devAlloc(&dev_iuvIdxMap, nSubVtx*sizeof(int2));
  hipMemcpy(dev_iuvIdxMap, iuvIdxMap, nSubVtx*sizeof(int2), hipMemcpyHostToDevice);
  delete uvIdxMap;
  delete iuvIdxMap;

  if (nSub > 2) {
    int2 *iuvInternalIdxMap = new int2[(nSub-1)*(nSub-2)/2];
    for (int v = 0; v <= nSub-3; v++) {
    for (int u = 0; u <= nSub-3-v; u++) {
      iuvInternalIdxMap[UV_IDX(u,v)] = make_int2(u+1,v+1);
    }}
    devAlloc(&dev_iuvInternalIdxMap, (nSub-1)*(nSub-2)*sizeof(int2)/2);
    hipMemcpy(dev_iuvInternalIdxMap, iuvInternalIdxMap, (nSub-1)*(nSub-2)*sizeof(int2)/2, hipMemcpyHostToDevice);
    delete iuvInternalIdxMap;
  }

  // d*(d-1)/2 - dmin*(dmin-1)/2
  fprintf(stderr, "creating patch data\n");
  devAlloc(&dev_bezPatch, nBasis2*nSubVtx*((degMax + 1)*degMax / 2 - degMin*(degMin - 1) / 2)*sizeof(float));
  devAlloc(&dev_wgtPatch, nSubVtx*((degMax + 1)*degMax / 2 - degMin*(degMin - 1) / 2)*sizeof(float));
  dim3 blkSize(16,16), blkCnt;
  blkCnt.x = (nDeg+blkSize.x-1)/blkSize.x;
  blkCnt.y = (nSubVtx + blkSize.y - 1) / blkSize.y;
  int nTessSM = (nBasis + 2) * blkSize.x * blkSize.y * sizeof(float2);
  for (int d = degMin; d <= degMax; d++) {
    int dOff = d*(d - 1) - degMin*(degMin - 1);
    dOff /= 2;
    kBezEval<<<blkCnt,blkSize,nTessSM>>>(d, nBasis, nSubVtx, dev_uvIdxMap, &dev_bezPatch[dOff*nSubVtx*nBasis2], &dev_wgtPatch[dOff*nSubVtx]);
    checkCUDAError("kBezEval", __LINE__);
  }
  
  fprintf(stderr, "creating sample data\n");
  if (canUseTexObjs) {
    genSampTex();
    checkCUDAError("genSampTex", __LINE__);
  }
}

void PPM::devTessInit() {
  fprintf(stderr, "creating edge tesselation\n");
  if (useVisualize) {
    size_t nBytes;
    hipGraphicsMapResources(1, &dev_vboTessIdx, 0);
    hipGraphicsResourceGetMappedPointer((void**)&dev_tessIdx, &nBytes, dev_vboTessIdx);
  } else {
    devAlloc(&dev_tessIdx, 3*nFace*nSubFace*sizeof(int));
  }
  
  dim3 blkSize(1024), blkCnt((nHe +1023) / 1024);
  kGetHeTessInfo<<<blkCnt, blkSize>>>(nHe, degMin, dev_heLoops, dev_heFaces, dev_vBndList);
  kGetHeTessInfo<<<blkCnt, blkSize>>>(nHe, degMin, dev_heLoops, dev_heLoops, dev_vBndList);


  devAlloc(&dev_heTessIdx, nHe*sizeof(int));
  kGetHeTessIdx<<<blkCnt, blkSize>>>(nHe, dev_heLoops, dev_heTessIdx);

  thrust::counting_iterator<int> order_itr(0);
  thrust::device_vector<int> order_vec(order_itr, order_itr+nHe), order2_vec(order_itr, order_itr+nHe);
  thrust::device_ptr<int> heTessIdx_ptr(dev_heTessIdx);
  thrust::sort_by_key(heTessIdx_ptr, heTessIdx_ptr+nHe, order_vec.begin());
  thrust::sort_by_key(order_vec.begin(), order_vec.end(), order2_vec.begin());
  hipMemcpy(dev_heTessIdx, order2_vec.data().get(), nHe*sizeof(int), hipMemcpyDeviceToDevice);
 
  blkSize.x = 64;
  blkSize.y = 16;
  blkCnt.x = (nFace + blkSize.x - 1) / blkSize.x;
  blkCnt.y = (nSubVtx + blkSize.y - 1) / blkSize.y;
  kTessEdges_Alt<<<blkCnt, blkSize>>>(nVtx, nHe, nFace, nSub, dev_heLoops, dev_heFaces, dev_heTessIdx, order2_vec.data().get(), dev_iuvIdxMap, dev_tessIdx);
  checkCUDAError("kTessEdges", __LINE__);
  
  if (useVisualize)
    hipGraphicsUnmapResources(1, &dev_vboTessIdx, 0);
  
  if (!useVisualize)
    devAlloc(&dev_tessVtx, PPM_NVARS*(nFace*(nSub-1)*(nSub-2)/2 + nHe*(nSub-1)/2 + nVtx)*sizeof(float));
  devAlloc(&dev_tessWgt, nFace*nSubVtx*sizeof(float));

}

// allocate and initialize PPM data
void PPM::devInit() {
  devMeshInit();
  devPatchInit();
  devCoeffInit();
  devTessInit();
}

void PPM::updateCoeff() {
  dim3 blkDim(16,64), blkCnt;
  blkCnt.x = (nBasis2 + blkDim.x - 1) / blkDim.x;
  blkCnt.y = (nVtx + blkDim.y - 1) / blkDim.y;
  kUpdateCoeff<<<blkCnt,blkDim>>>(nBasis2, nVtx, bezier->dev_V, 1.0, dev_dv, dev_coeff, 0.1f);
  checkCUDAError("kUpdateCoeff", __LINE__);
}

float PPM::update() {
  if (!isBuilt)
    return 0.0f;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
  hipEventRecord(start, 0);

  dim3 blkDim;
  dim3 blkCnt;

  // generate/update bezier coefficients
  updateCoeff();

  // calculate new vertex positions
  if (useVisualize) {
    size_t nBytes;
    hipGraphicsMapResources(1, &dev_vboTessVtx, 0);
    hipGraphicsResourceGetMappedPointer((void**)&dev_tessVtx, &nBytes, dev_vboTessVtx);
  }
  /*
  if (useTessSM) {
    hipMemset(dev_tessVtx, 0, PPM_NVARS * nFace*nSubVtx*sizeof(float));
    hipMemset(dev_tessWgt, 0, nFace*nSubVtx*sizeof(float));

    blkDim.x = 8;
    blkDim.y = 16;
    blkDim.z = 8;
    blkCnt.x = (nHe + blkDim.x - 1) / blkDim.x;
    blkCnt.y = (nSubVtx + blkDim.y - 1) / blkDim.y;
    blkCnt.z = (PPM_NVARS + blkDim.z - 1) / blkDim.z;
    int smSize = (blkDim.z * blkDim.x * nBasis2) * sizeof(float);
    kTessVtxSM<<<blkCnt, blkDim, smSize>>>(nVtx, nHe, nSub, nSubVtx, nBasis2, degMin,
        dev_heFaces, dev_bezPatch, dev_wgtPatch, dev_coeff, dev_iuvIdxMap, dev_tessVtx, dev_tessWgt);
    checkCUDAError("kTessVtxSM", __LINE__); 

    blkDim.x = 32;
    blkDim.y = 32;
    blkDim.z = 1;
    blkCnt.x = (nFace + blkDim.x - 1) / blkDim.x;
    blkCnt.y = (nSubVtx + blkDim.y - 1) / blkDim.y;
    blkCnt.z = 1;
    kWeightScale<<<blkCnt, blkDim>>>(nFace, nSubVtx, dev_tessVtx, dev_tessWgt);
    checkCUDAError("kWeightScale", __LINE__);
  } else {
    blkDim.x = 32;
    blkDim.y = 4;
    blkDim.z = 4;
    blkCnt.x = (nFace + blkDim.x - 1) / blkDim.x;
    blkCnt.y = (nSubVtx + blkDim.y - 1) / blkDim.y;
    blkCnt.z = (PPM_NVARS + blkDim.z - 1) / blkDim.z;
    hipMemset(dev_tessVtx, 0, PPM_NVARS * nFace*nSubVtx*sizeof(float));
    kTessVtx<<<blkCnt, blkDim>>>(nVtx, nFace, nSub, nSubVtx, nBasis2, degMin,
      dev_heFaces, dev_bezPatch, dev_wgtPatch, dev_coeff, dev_iuvIdxMap, dev_tessVtx);
    checkCUDAError("kTessVtx", __LINE__);
  }
  */
  hipMemset(dev_tessVtx, 0, PPM_NVARS * (nFace*(nSub-1)*(nSub-2)/2 + nHe*(nSub-1)/2 + nVtx) * sizeof(float));
  
  if (nSub > 2) {
    blkDim.x = 16;
    blkDim.y = 16;
    blkDim.z = 4;
    blkCnt.x = (nFace + blkDim.x - 1) / blkDim.x;
    blkCnt.y = ((nSub-1)*(nSub-2)/2 + blkDim.y - 1) / blkDim.y;
    blkCnt.z = (PPM_NVARS + blkDim.z - 1) / blkDim.z;
    kTessVtx_Face<<<blkCnt, blkDim>>>(nVtx, nFace, nSub, nBasis2,
      dev_heFaces, dev_bezPatch, dev_wgtPatch, dev_coeff, dev_iuvInternalIdxMap, dev_tessVtx);
    checkCUDAError("kTessVtx_Face", __LINE__);
  }
  
  if (nSub > 1) {
    blkDim.x = 64;
    blkDim.y = 4;
    blkDim.z = 4;
    blkCnt.x = (nHe/2 + blkDim.x - 1) / blkDim.x;
    blkCnt.y = (nSub-1 + blkDim.y - 1) / blkDim.y;
    blkCnt.z = (PPM_NVARS + blkDim.z - 1) / blkDim.z;
    kTessVtx_Edge_Alt<<<blkCnt, blkDim>>>(nVtx, nHe, nSub, nBasis2,
      dev_heLoops, dev_heTessIdx, dev_bezPatch, dev_wgtPatch, dev_coeff, dev_tessVtx + PPM_NVARS*nFace*(nSub-1)*(nSub-2)/2);
    checkCUDAError("kTessVtx_Edge", __LINE__);
  }
  
  blkDim.x = 64;
  blkDim.y = 16;
  blkDim.z = 1;
  blkCnt.x = (nVtx + blkDim.x - 1) / blkDim.x;
  blkCnt.y = (PPM_NVARS + blkDim.y - 1) / blkDim.y;
  blkCnt.z = 1;
  kTessVtx_Vtx<<<blkCnt, blkDim>>>(nVtx, nSub, nBasis2,
    dev_heLoops, dev_vBndList, dev_bezPatch, dev_wgtPatch, dev_coeff, dev_tessVtx + PPM_NVARS*(nFace*(nSub-1)*(nSub-2)/2 + nHe*(nSub-1)/2));
  checkCUDAError("kTessVtx_Vtx", __LINE__);
  
  if (useVisualize)
    hipGraphicsUnmapResources(1, &dev_vboTessVtx, 0);

  float dt;
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&dt, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return dt;
}

// free dcel data
void PPM::devFree() {
  for (void *p : allocList)
    hipFree(p);
  allocList.clear();

  delete bezier;
}

